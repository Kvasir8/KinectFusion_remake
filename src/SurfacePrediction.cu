#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "SurfaceReconstruction.cuh"
#include "PointCloud.h"


//voxweights vox values, depthmap, camparams needs to be copied needs to be included
__global__ void surfacePredictionKernel(Vector3d min, Vector3d max, double* voxWeights, double* voxValues, Matrix4f currentCameraPose, Matrix4f transMatrixcur,
	Vector3f* points, Vector3f* normals, float* camparams, Vector3f voxelDistance)
{
	//Pixel coordinates
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	int respectiveX = int(tid % 640); // Column in image pixels
	int respectiveY = int(tid / 640); // Row in image pixels
	//start with the position of the camera which is equal to the translation in currentCameraPose
	Vector2i pixelCoord = Vector2i(respectiveX, respectiveY);
	//Now to cam coordinates

	Vector3f pixelsinCamSpace = Vector3f((pixelCoord[0] - camparams[2]) / camparams[0],
										(pixelCoord[1] - camparams[3]) / camparams[1],
										1.0f);

	// A ray with direction in World Coordinates
	Vector3f raydirection = transMatrixcur.block<3, 3>(0, 0) * pixelsinCamSpace;
	//could not write it in one line do not know why the direction seems to be updating nicely, seems to be updating without return
	raydirection.normalize();

	float updateStepRay = voxelDistance[2];
	Vector3f raystart;
	Vector3f rayprev;
	//to world coordinates in grid
	raystart = (transMatrixcur.block<3, 1>(0, 3) + raydirection * updateStepRay )/ voxelDistance[2];
	rayprev = raystart;
	float previousTsdf = 1;
	float tsdf = 1;

	for (int i = 0; i < 512 ; i++) {
		//ray is in the grid coord
		raystart = (transMatrixcur.block<3, 1>(0, 3) + (raydirection * updateStepRay)) / voxelDistance[2];
		
		if(int(raystart.x())>0 && int(raystart.y())>0 && int(raystart.z())>0 &&
			(raystart.x()) < 512 && int(raystart.y()) < 512 && int(raystart.z()) <512 &&
			voxValues[int(raystart.x()) * 512 * 512 + int(raystart.y()) * 512 + int(raystart.z())]!=1 )
		{
			previousTsdf = tsdf;
			tsdf = voxValues[int(raystart.x()) * 512 * 512 + int(raystart.y()) * 512 + int(raystart.z())];
			
			if (tsdf <= 0 && previousTsdf > 0) {
				Vector3f pointFound;
				//grid to world
				//update step ray is the length of the raydirection
				//this seems to be working
				
				raystart = rayprev + raydirection * previousTsdf / (previousTsdf-tsdf);
				pointFound = Vector3f(min[0] + voxelDistance[0] * raystart[0],
									  min[1] + voxelDistance[1] * raystart[1],
									  min[2] + voxelDistance[2] * raystart[2]);
				
				//	printf(" PointFound, Raystart: %f %f %f , % i, % i, % i \n",  pointFound[0], pointFound[1], pointFound[2], int(raystart.x()), int(raystart.y()), int(raystart.z()));
				
				points[pixelCoord[1] * 640 + pixelCoord[0]] = currentCameraPose.block<3,3>(0,0)* pointFound + currentCameraPose.block<3,1>(0,3);

				//Now we need to find the normals
				/* 
				*	How the interpolation works :
				*	Look at voxels upper lower, right left, front behind, and find out the value 
				*	that normal should have by interpolating them
				*/
				break;

			}
		}
		rayprev = raystart;
		updateStepRay += voxelDistance[2];

	}
	//can never jump over a voxel this way
	//goingto update through z axis therefore I chose the distance along the z axis
	
}
namespace CUDA {
	//Also need the spacing to be able to project the voxels to ->World->cam->image plane, will be calculated on cuda but needs to be adressed for faster update
	//added min and max point of the voxel, min left lower corner, max= left uppercorner
	void SurfacePrediction(Vector3d& min, Vector3d& max, double* voxWeights, double* voxValues, Matrix4f& currentCameraPose, Matrix4f& transMatrixcur,
		std::vector<Vector3f>& points, std::vector<Vector3f>& normals, std::vector<float>& camparams, PointCloud& frame){
		double* voxWeightPointer;
		double* voxValuePointer;

		float* camparamPointer; //params of the source
		Vector3f* pointsPointer;
		Vector3f* normalsPointer;
		//Mallocs
		//Each has a value for one voxel.
		//hipMalloc(&voxWeightPointer, sizeof(double) * 512 * 512 * 512);
		hipMalloc(&voxValuePointer, sizeof(double) * 512 * 512 * 512);
		//4 variables in camparams Look at exercise 5 for multiplication
		hipMalloc(&camparamPointer, sizeof(float) * 4);
		hipMalloc((void**)&pointsPointer, sizeof(Vector3f) * 640 * 480);
		hipMalloc((void**)&normalsPointer, sizeof(Vector3f) * 640 * 480);



		//CudaHostalloc should be used can be changed later, I dont want anything to get crashed
		//hipMemcpy(voxWeightPointer, voxWeights, sizeof(double) * 512 * 512 * 512, hipMemcpyHostToDevice);
		hipMemcpy(voxValuePointer, voxValues, sizeof(double) * 512 * 512 * 512, hipMemcpyHostToDevice);
		hipMemcpy(camparamPointer, camparams.data(), sizeof(float) * 4, hipMemcpyHostToDevice);
		hipMemcpy(pointsPointer, points.data(), sizeof(Vector3f) * 640 * 480, hipMemcpyHostToDevice);
		hipMemcpy(normalsPointer, normals.data(), sizeof(Vector3f) * 640 * 480, hipMemcpyHostToDevice);

		//Now everything works with copying should create a grid, block and threads to be able to iterate over the values and weight in cuda
		// Same thing as the Pose Estimation, We have x512 y512 z512 weights and values. we are only going to update it by depth meaning
		//First start with for i in range(z): update where 512 512 x and y think of it as an image by 512 512. We are going to update by looking behind of the voxel
		//In the iteration. So 512=2^9 => 512*8, 512/8 => 4096, 64. This should be generalized and not be calcualted by hand!!!!
		Vector3f distanceBetweenVoxels((max[0] - min[0]) / 511,
										(max[1] - min[1]) / 511,
										(max[2] - min[2]) / 511);
		surfacePredictionKernel << <4800, 64 >> > (min, max, voxWeightPointer, voxValuePointer, currentCameraPose, transMatrixcur, pointsPointer, normalsPointer, camparamPointer,distanceBetweenVoxels);

		//hipMemcpy(voxWeights, voxWeightPointer, sizeof(double) * 512 * 512 * 512, hipMemcpyDeviceToHost);
		//hipMemcpy(voxValues, voxValuePointer, sizeof(double) * 512 * 512 * 512, hipMemcpyDeviceToHost);
		hipMemcpy(frame.m_points.data(), pointsPointer, sizeof(Vector3f) * 640 * 480, hipMemcpyDeviceToHost);
		hipMemcpy(frame.m_normals.data(), normalsPointer, sizeof(Vector3f) * 640 * 480, hipMemcpyDeviceToHost);


		//call the kernel here
		hipDeviceSynchronize();
		//hipFree(voxWeightPointer);
		hipFree(voxValuePointer);
		hipFree(camparamPointer);
		hipFree(pointsPointer);
		hipFree(normalsPointer);

		//surfaceReconstructionKernel << <4096, 64 >> > (min, max, voxWeightPointer, voxValuePointer, currentCameraPose, transMatrixcur, depthMapPointer, normalsPointer, camparamPointer);


	}
}
