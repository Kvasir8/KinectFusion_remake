#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "SurfaceReconstruction.cuh"

//voxweights vox values, depthmap, camparams needs to be copied needs to be included
__global__ void surfaceReconstructionKernel(Vector3d min, Vector3d max, double* voxWeights, double* voxValues, Matrix4f currentCameraPose, Matrix4f transMatrixcur,
	float* depthMap,Vector3f* normals, float* camparams)
{
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	// (max[0] - min[0]) / (dx - 1)
	//totaldistance with respect to x y z
	//this can be included in Cuda call
	if (tid<512*512) {
		Vector3f distanceBetweenVoxels((max[0] - min[0]) / 511,
										(max[1] - min[1]) / 511,
										(max[2] - min[2]) / 511);
		int respectiveX = int(tid / 512); // Row in image
		int respectiveY = int(tid % 512); // Column in image

		//Voxel in World coordinates
		Vector3f locationVoxelG;
		//Voxel in camSpace
		Vector3f locationVoxelC;
		//Voxel in pixels
		Vector2i pixelCoord;
		int counter = 0;

		//iterate over the depth, look behind of the voxel!!!!!!!
		for (int z = 0; z < 512; z++) {
			//find the location of the voxel in worldspace, Seems to be working fine
			locationVoxelG = Vector3f(min[0] + distanceBetweenVoxels[0] * respectiveX,
								      min[1] + distanceBetweenVoxels[1] * respectiveY,
									  min[2] + distanceBetweenVoxels[2] * z);
			//Now go to the camspace of the given PointCloud
			locationVoxelC = currentCameraPose.block<3, 3>(0, 0) * locationVoxelG + currentCameraPose.block<3, 1>(0, 3);
			//bring it to image Coordinates project it same as the pose estimation
			pixelCoord=Vector2i( int(locationVoxelC[0] * camparams[0] / locationVoxelC[2] + camparams[2]),
						int(locationVoxelC[1] * camparams[1] / locationVoxelC[2] + camparams[3]));
			
			if (pixelCoord[0] > 0 && pixelCoord[0] < 640 && pixelCoord[1] > 0 && pixelCoord[1] < 480) {
				//voxWeights[0] = 10000;
				//From the original paper Kinectfusion equations 6 7 8 9
				float lambda =(locationVoxelC / locationVoxelC[2]).norm();
				Vector3f translation= currentCameraPose.block<3, 1>(0, 3);
				float RawDepth = depthMap[pixelCoord[1] * 640 + pixelCoord[0]];
				float voxCurrentValue;
				if (RawDepth != MINF) {
					float sdf = (translation - locationVoxelG).norm() - RawDepth;
					//Voila
					//To measure free space
					Vector3f freeSpace = locationVoxelG / locationVoxelG.norm();
					/*From paper cos=> The associated weight WRk(p) is proportional
						to cos(q) / Rk(x), where q is the angle between the associated
						pixel ray directionand the surface normal measurement in the local
						frame.*/
					float cosangle = freeSpace.dot(normals[pixelCoord[1] * 640 + pixelCoord[0]])/ (freeSpace.norm() * normals[pixelCoord[1] * 640 + pixelCoord[0]].norm());
					if (sdf > 0) {
						voxCurrentValue = std::min(1.0f, sdf);
						//printf("sdf: %f", sdf);
						if (voxCurrentValue < 1) {
							//printf("sdf: %f", voxCurrentValue);
						}
					}
					else {
						voxCurrentValue = std::max(-1.0f, sdf);
					}
					//x* dy* dz + y * dz + z;
					//assuming equal weights we can update it later// running average added
					voxValues[respectiveX * 512 * 512 + respectiveY * 512 + z] = (voxWeights[respectiveX * 512 * 512 + respectiveY * 512 + z]*voxValues[respectiveX * 512 * 512 + respectiveY * 512 + z] 
																				+ voxCurrentValue)/(voxWeights[respectiveX * 512 * 512 + respectiveY * 512 + z]+1);
					//This should be updated
					voxWeights[respectiveX * 512 * 512 + respectiveY * 512 + z]+= 1;// voxWeights[respectiveX * 512 * 512 + respectiveY * 512 + z];
				}
			}
		}
		//test
		


	}
}
namespace CUDA{
	//Also need the spacing to be able to project the voxels to ->World->cam->image plane, will be calculated on cuda but needs to be adressed for faster update
	//added min and max point of the voxel, min left lower corner, max= left uppercorner
	void SurfaceReconstruction(Vector3d& min, Vector3d& max, double* voxWeights, double* voxValues, Matrix4f& currentCameraPose, Matrix4f& transMatrixcur,
		float* depthMap, std::vector<Vector3f>& normals, std::vector<float>& camparams) {
		//calling the kernel
		double* voxWeightPointer;
		double* voxValuePointer;

		float* camparamPointer; //params of the source
		float* depthMapPointer;
		Vector3f* normalsPointer;
		//Mallocs
		//Each has a value for one voxel.
		hipMalloc(&voxWeightPointer, sizeof(double) * 512 * 512 * 512);
		hipMalloc(&voxValuePointer, sizeof(double) * 512 * 512 * 512);
		//4 variables in camparams Look at exercise 5 for multiplication
		hipMalloc(&camparamPointer, sizeof(float) * 4);
		hipMalloc(&depthMapPointer, sizeof(float) * 640 * 480);
		hipMalloc((void**)&normalsPointer, sizeof(Vector3f) * 640 * 480);



		//CudaHostalloc should be used can be changed later, I dont want anything to get crashed
		hipMemcpy(voxWeightPointer, voxWeights, sizeof(double) * 512 * 512 * 512, hipMemcpyHostToDevice);
		hipMemcpy(voxValuePointer, voxValues, sizeof(double) * 512 * 512 * 512, hipMemcpyHostToDevice);
		hipMemcpy(camparamPointer, camparams.data(), sizeof(float) * 4, hipMemcpyHostToDevice);
		hipMemcpy(depthMapPointer, depthMap, sizeof(float) * 640*480, hipMemcpyHostToDevice);
		hipMemcpy(normalsPointer, normals.data(), sizeof(Vector3f) * 640 * 480, hipMemcpyHostToDevice);

		//Now everything works with copying should create a grid, block and threads to be able to iterate over the values and weight in cuda
		// Same thing as the Pose Estimation, We have x512 y512 z512 weights and values. we are only going to update it by depth meaning
		//First start with for i in range(z): update where 512 512 x and y think of it as an image by 512 512. We are going to update by looking behind of the voxel
		//In the iteration. So 512=2^9 => 512*8, 512/8 => 4096, 64. This should be generalized and not be calcualted by hand!!!!
		surfaceReconstructionKernel << <4096, 64 >> > (min,max,voxWeightPointer, voxValuePointer, currentCameraPose, transMatrixcur, depthMapPointer, normalsPointer, camparamPointer);

		hipMemcpy(voxWeights, voxWeightPointer, sizeof(double) * 512 * 512 * 512, hipMemcpyDeviceToHost);
		hipMemcpy(voxValues, voxValuePointer, sizeof(double) * 512 * 512 * 512, hipMemcpyDeviceToHost);


		//call the kernel here
		hipDeviceSynchronize();
		hipFree(voxWeightPointer);
		hipFree(voxValuePointer);
		hipFree(camparamPointer);
		hipFree(depthMapPointer);

	}
}
