#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "SurfaceMeasurement.cuh"
#include <stdio.h>
#include <iostream>
#include <time.h>


//The GPU does the work
__global__ void vectorAdd(const int*  a, const int*  b, int*  c) {

	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid < 8){
		c[tid]= a[tid]+ b[tid];
	}
}
__global__ void initSensorFrame_kernel(const float* depthMap, const Matrix3f rotationInv, const Vector3f translationInv,
	 float* camparams, Vector3f* pointsTmp) {
	//this should be done on gpu meaning 640 as input
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;	
	if (tid < 640*480) { //640*480 being the height and width
		if (depthMap[tid] == MINF) {
			pointsTmp[tid] = Vector3f(MINF, MINF, MINF);
		}
		else {
			int u = (blockIdx.x * blockDim.x + threadIdx.x)%640;
			int v = int((blockIdx.x * blockDim.x + threadIdx.x) / 640);
			//Camera Intrincs ~=~ camparams
			pointsTmp[tid] = Vector3f((u - camparams[2]) * depthMap[tid] / camparams[0], (v - camparams[3]) * depthMap[tid] / camparams[1] , depthMap[tid]);
		}
	}
}

__global__ void normalMap_kernel(const Vector3f* pointsTmp, float maxDistanceHalved, Vector3f* normalsTmp) {
	//this should be done on gpu meaning 640 as input
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	//modulo operator for edge cases in the border
	if (tid < 640 * 480 || (tid%640)%480!=0 ) { //640*480 being the height and width
		const Vector3f du =   (pointsTmp[tid + 1] - pointsTmp[tid - 1]);
		const Vector3f dv =   (pointsTmp[tid + 640] - pointsTmp[tid - 640]);
		if (du.norm() == MINF || dv.norm() == MINF) {
			normalsTmp[tid] = Vector3f(MINF, MINF, MINF);
		}
		else {
			normalsTmp[tid] = du.cross(dv);
			normalsTmp[tid].normalize();
		}
	}
}
	
	
__global__ void example_kernel(float* depthmap,Vector3f* pointsTmp) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid < 640 * 480)
	{
		
		float depth = depthmap[tid];
		pointsTmp[tid] = Vector3f(depth, depth, depth);
	}

}



namespace CUDA {
	//Wrapper to call the kernel function on the GPU

	void initSensorFrame(float depthMap[], Matrix3f& rotationInv, Vector3f& translationInv,
		std::vector<float>& camparams, std::vector<Vector3f>& pointsTmp) {

		//size_t bytes = sizeof(float) * 4+ sizeof(Vector3f) * (sizeof(pointsTmp) + 1)+ sizeof(Matrix3f);//+1 because translation

		//allocate memory on the GPU of the size you want to change in our case sizeof(int) * N;
		float* depthPointer;
		float* camparamPointer;
		Vector3f* pointsPointer;

		hipMalloc(&depthPointer, sizeof(float) * 640 * 480);
		//4 variables in camparams
		hipMalloc(&camparamPointer, sizeof(float) * 4);
		hipMalloc((void**)&pointsPointer, sizeof(Vector3f) * 640 * 480);

		//copy the data to the GPU
		hipMemcpy(depthPointer, depthMap, sizeof(float) * 640 * 480, hipMemcpyHostToDevice);
		hipMemcpy(camparamPointer, camparams.data(), sizeof(float) * 4, hipMemcpyHostToDevice);
		hipMemcpy(pointsPointer, pointsTmp.data(), sizeof(Vector3f) * 640 * 480, hipMemcpyHostToDevice);
		
		//8 threads 1 block 640*480 
		initSensorFrame_kernel << <4800, 64 >> > (depthPointer, rotationInv, translationInv, camparamPointer, pointsPointer);
		//After the calculation copy the value back to the CPU 
		
		hipMemcpy(pointsTmp.data(), pointsPointer, sizeof(Vector3f) * 640 * 480, hipMemcpyDeviceToHost);

		//hipDeviceSynchronize();
		//Free the allocated memory on the GPU
		hipFree(depthPointer);
		hipFree(camparamPointer);
		hipFree(pointsPointer);

	}
	void initnormalMap(std::vector<Vector3f>& pointsTmp, float maxDistanceHalved, std::vector<Vector3f>& normalsTmp){

		Vector3f* pointsPointer;
		Vector3f* normalsPointer;

		hipMalloc((void**)&pointsPointer, sizeof(Vector3f) * 640 * 480);
		hipMalloc((void**)&normalsPointer, sizeof(Vector3f) * 640 * 480);

		hipMemcpy(pointsPointer, pointsTmp.data(), sizeof(Vector3f) * 640 * 480, hipMemcpyHostToDevice);
		hipMemcpy(normalsPointer, normalsTmp.data(), sizeof(Vector3f) * 640 * 480, hipMemcpyHostToDevice);

		normalMap_kernel<<<4800, 64>>> (pointsPointer, maxDistanceHalved,normalsPointer);

		hipMemcpy(normalsTmp.data(), normalsPointer, sizeof(Vector3f) * 640 * 480, hipMemcpyDeviceToHost);

		hipFree(pointsPointer);
		hipFree(normalsPointer);
	}


	//just example tried here to find the bug XD
	void example(float depthMap[], std::vector<Vector3f>& pointsTmp) {
		float* depthPointer;
		Vector3f* pointsPointer;

		hipMalloc(&depthPointer, sizeof(float) * 640 * 480);
		hipMalloc((void**)&pointsPointer, sizeof(Vector3f) * 640 * 480);

		hipMemcpy(depthPointer, depthMap, sizeof(float) * 640 * 480, hipMemcpyHostToDevice);
		hipMemcpy(pointsPointer, pointsTmp.data(), sizeof(Vector3f) * 640 * 480, hipMemcpyHostToDevice);
		
		example_kernel <<< 4800, 64 >>>(depthPointer,pointsPointer);
		hipMemcpy(pointsTmp.data(), pointsPointer, sizeof(Vector3f) * 640 * 480, hipMemcpyDeviceToHost);

		hipFree(pointsPointer);
		hipFree(depthPointer);

	}


}